#include <hip/hip_runtime.h>
#include "utils.hpp"
#include "naive-kernel.cuh"

int main() {
	constexpr unsigned int M = 6; // Height of A and C
	constexpr unsigned int N = 8; // Width of B and C
	constexpr unsigned int K = 4; // Width of A and Height of B
	constexpr unsigned int BLOCK_SIZE = 32; // Block size for CUDA kernel
	constexpr unsigned int GRID_SIZE_X = CEIL_DIV(N, BLOCK_SIZE);
	constexpr unsigned int GRID_SIZE_Y = CEIL_DIV(M, BLOCK_SIZE);

	float* d_A, * d_B, * d_C;
	{
		constexpr size_t A_size = K * M * sizeof(float);
		float* h_A = generateMatrix(K, M);
		printMatrix<M, K>(h_A, M, K);
		hipMalloc(reinterpret_cast<void**>(&d_A), A_size);
		hipMemcpy(d_A, h_A, A_size, hipMemcpyHostToDevice);
		delete[] h_A;
		std::cout << '\n';
	}
	{
		constexpr size_t B_size = K * N * sizeof(float);
		float* h_B = generateMatrix(K, N);
		printMatrix<K, N>(h_B, K, N);
		hipMalloc(reinterpret_cast<void**>(&d_B), B_size);
		hipMemcpy(d_B, h_B, B_size, hipMemcpyHostToDevice);
		delete[] h_B;
		std::cout << '\n';
	}
	{
		constexpr size_t C_size = M * N * sizeof(float);
		float* h_C = zeroMatrix<float>(M, N);
		hipMalloc(reinterpret_cast<void**>(&d_C), C_size);
		hipMemcpy(d_C, h_C, C_size, hipMemcpyHostToDevice);
		delete[] h_C;
	}
	// Initialize matrices A, B, and C

	const dim3 blockdim(32, 32);
	const dim3 griddim(GRID_SIZE_X, GRID_SIZE_Y);

	naiveSGEMM<M, N, K> << <griddim, blockdim >> > (d_A, d_B, d_C);

	{
		constexpr size_t C_size = M * N * sizeof(float);
		float* h_C = new float[M * N];
		hipMemcpy(h_C, d_C, C_size, hipMemcpyDeviceToHost);
		printMatrix<M, N>(h_C, M, N, 10U, 3U);
		delete[] h_C;
		std::cout << '\n';

	}
	hipFree(reinterpret_cast<void*>(d_A));
	hipFree(reinterpret_cast<void*>(d_B));
	hipFree(reinterpret_cast<void*>(d_C));

	return 0;
}
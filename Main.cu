#include <hip/hip_runtime.h>
#include "utils.hpp"
#include "naive-kernel.cuh"

int main() {
	constexpr unsigned int M = 4; // Height of A and C
	constexpr unsigned int N = 4; // Width of B and C
	constexpr unsigned int K = 4; // Width of A and Height of B

	float* d_A, * d_B, * d_C;
	{
		constexpr size_t A_size = K * M * sizeof(float);
		float* h_A = generateMatrix(K, M);
		hipMalloc((void**)&d_A, A_size);
		hipMemcpy(d_A, h_A, A_size, hipMemcpyHostToDevice);
		delete[] h_A;
	}
	{
		constexpr size_t B_size = K * N * sizeof(float);
		float* h_B = generateMatrix(K, N);
		hipMalloc((void**)&d_B, B_size);
		hipMemcpy(d_B, h_B, B_size, hipMemcpyHostToDevice);
		delete[] h_B;
	}
	{
		constexpr size_t C_size = M * N * sizeof(float);
		float* h_C = zeroMatrix<float>(M, N);
		hipMalloc((void**)&d_C, C_size);
		hipMemcpy(d_C, h_C, C_size, hipMemcpyHostToDevice);
		delete[] h_C;
	}
	// Initialize matrices A, B, and C

	const dim3 blockdim(32, 32);
	const dim3 griddim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));

	naiveSGEMM<M, N, K> << <griddim, blockdim >> > (d_A, d_B, d_C);


	return 0;
}
#include <hip/hip_runtime.h>
#include "utils.hpp"
#include "kernel-2-1D.cuh"
#include <hipblas.h>

//#define iscuBLAS 1

// Add this helper function at the top of your file after the includes
inline void checkCudaError(hipError_t error, const char* function, const char* file, int line) {
	if (error != hipSuccess) {
		std::cerr << "CUDA error in " << file << ":" << line << " (" << function << ") : "
			<< hipGetErrorString(error) << std::endl;
		// Ensure all CUDA work is terminated
		hipDeviceReset();
		exit(EXIT_FAILURE);
	}
}

// Macro to make it easier to call the error checker
#define CUDA_CHECK(x) checkCudaError(x, #x, __FILE__, __LINE__)

constexpr bool iscuBLAS = false; // Set to true to use cuBLAS, false to use custom SGEMM kernel
constexpr bool checkIfWorks = true;	// Set to true to check if the SGEMM works correctly by comparing it with cuBLAS
constexpr uint32_t dim = 2048U;	// Size of the matrices (dim x dim)

int main() {
	constexpr uint32_t M = dim; // Height of A and C
	constexpr uint32_t N = dim; // Width of B and C
	constexpr uint32_t K = dim; // Width of A and Height of B
	constexpr uint32_t BLOCK_SIZE = 32U; // Block size for CUDA kernel
	constexpr uint32_t GRID_SIZE_X = CEIL_DIV(N, BLOCK_SIZE);
	constexpr uint32_t GRID_SIZE_Y = CEIL_DIV(M, BLOCK_SIZE);
	constexpr size_t A_size = M * K * sizeof(float);
	constexpr size_t B_size = K * N * sizeof(float);
	constexpr size_t C_size = M * N * sizeof(float);

	const float alpha = 1.0f;
	const float beta = 0.0f;

	float* d_A, * d_B, * d_C;
	hipMalloc(reinterpret_cast<void**>(&d_A), A_size);
	hipMalloc(reinterpret_cast<void**>(&d_B), B_size);
	hipMalloc(reinterpret_cast<void**>(&d_C), C_size);

	float* h_A = generateMatrix(M, K, -10.0f, 10.0f);
	float* h_B = generateMatrix(K, N, -10.0f, 10.0f);
	float* h_C = zeroMatrix<float>(M, N);

	hipMemcpy(reinterpret_cast<void*>(d_A), h_A, A_size, hipMemcpyHostToDevice);
	hipMemcpy(reinterpret_cast<void*>(d_B), h_B, B_size, hipMemcpyHostToDevice);
	hipMemcpy(reinterpret_cast<void*>(d_C), h_C, C_size, hipMemcpyHostToDevice);

	delete[] h_A;
	delete[] h_B;
	delete[] h_C;

	if constexpr(iscuBLAS) {
		hipblasHandle_t handle;
		hipblasCreate(&handle);

		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_A, M, d_B, K, &beta, d_C, M);
		hipDeviceSynchronize();
		hipblasDestroy(handle);
		std::cout << "cuBLAS SGEMM finished\n";
	}

	#ifdef OneDimensional
		const dim3 blockDim(BLOCK_SIZE * BLOCK_SIZE);
		const dim3 griddim(GRID_SIZE_X, GRID_SIZE_Y);
		SGEMM<M, N, K, BLOCK_SIZE> <<<griddim, blockDim >>> (d_A, d_B, d_C);
		CUDA_CHECK(hipDeviceSynchronize()); // Ensure the kernel has finished executing
		std::cout << "SGEMM finished with grid size: " << griddim.x << " * " << griddim.y << " and block size: " << blockDim.x << std::endl;
	#endif

	#ifdef TwoDimensional
		const dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
		const dim3 griddim(GRID_SIZE_X, GRID_SIZE_Y);
		SGEMM<M, N, K> << <griddim, blockDim >> > (d_A, d_B, d_C);
		hipDeviceSynchronize(); // Ensure the kernel has finished executing
		std::cout << "SGEMM finished with grid size: " << griddim.x << " * " << griddim.y << " and block size: " << blockDim.x << " * " << blockDim.y << std::endl;
	#endif
	
	// Disable if profiling with nsight
	if constexpr(checkIfWorks) {

		float* h_C1 = new float[M * N];	// Holds the reult of the checked SGEMM
		float* h_C2 = zeroMatrix<float>(M, N); // Holds the result of the naive SGEMM - thorougly checked so I can assume it is correct

		CUDA_CHECK(hipMemcpy(h_C1, d_C, C_size, hipMemcpyDeviceToHost)); // Copy the result of the SGEMM to host memory
		CUDA_CHECK(hipMemcpy(d_C, h_C2, C_size, hipMemcpyHostToDevice)); // Reset the device memory for the naive SGEMM

		hipblasHandle_t handle;
		hipblasCreate(&handle);

		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_A, M, d_B, K, &beta, d_C, M);
		hipDeviceSynchronize();
		hipblasDestroy(handle);
		
		std::cout << "cuBLAS SGEMM finished" << std::endl;
		hipMemcpy(h_C2, d_C, C_size, hipMemcpyDeviceToHost);

		// Check if they are equal and account for floating point precision
		if (AreEqualMatrices<M, N>(h_C1, h_C2, 2.5f)) {	
			std::cout << "The matrices are equal.\n";
		} else {
			std:: cout << "The matrices are NOT equal.\n" << "Matrix 1: " << std::endl;
			printMatrix<M, N>(h_C1, 10, 10, 15, 2);
			std::cout << "\nMatrix 2: " << std::endl;
			printMatrix<M, N>(h_C2, 10, 10, 15, 2);
		}

		delete[] h_C1;
		delete[] h_C2;

	}

	// Free device memory
	hipFree(reinterpret_cast<void*>(d_A));
	hipFree(reinterpret_cast<void*>(d_B));
	hipFree(reinterpret_cast<void*>(d_C));

	return 0;
}